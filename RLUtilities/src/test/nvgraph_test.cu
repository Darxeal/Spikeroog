#include <stdlib.h>
#include <inttypes.h>
#include <stdio.h>

#include <string>
#include <vector>

#include "misc/io.h"
#include "misc/timer.h"
#include "misc/graph.h"

#include "hip/hip_runtime.h"
#include "nvgraph.h"

std::string filename("C:\\Users\\sam\\Desktop\\LUT\\tuples.bin");

Graph g;

timer stopwatch;

void check(nvgraphStatus_t status) {
  if (status != NVGRAPH_STATUS_SUCCESS) {
    printf("ERROR : %d\n",status);
    exit(0);
  }
}

void nvgraph_spmv_test() {

  size_t vertex_numsets = 2;
  size_t edge_numsets = 1;
  float alpha = 1.0, beta = 1.0;
  void *alpha_p = (void *)&alpha, *beta_p = (void *)&beta;
  void** vertex_dim;
  hipDataType edge_dimT = HIP_R_32F;
  hipDataType* vertex_dimT;
  // nvgraph variables
  nvgraphHandle_t handle;
  nvgraphGraphDescr_t graph;
  nvgraphCSRTopology32I_t CSR_input;
  // Init host data
  vertex_dim  = (void**)malloc(vertex_numsets*sizeof(void*));
  vertex_dimT = (hipDataType*)malloc(vertex_numsets*sizeof(hipDataType));
  CSR_input = (nvgraphCSRTopology32I_t) malloc(sizeof(struct nvgraphCSRTopology32I_st));
  
  std::vector < float > x_h(g.num_vertices);
  std::vector < float > y_h(g.num_vertices);
  vertex_dim[0] = (void*)(&x_h[0]); 
  vertex_dim[1] = (void*)(&y_h[0]);
  vertex_dimT[0] = HIP_R_32F; 
  vertex_dimT[1] = HIP_R_32F;

  check(nvgraphCreate(&handle));
  check(nvgraphCreateGraphDescr(handle, &graph));

  CSR_input->nvertices = g.num_vertices; 
  CSR_input->nedges = g.num_edges;
  CSR_input->source_offsets = &g.offsets[0];
  CSR_input->destination_indices = &g.destinations[0];
  // Set graph connectivity and properties (tranfers)
  check(nvgraphSetGraphStructure(handle, graph, (void*)CSR_input, NVGRAPH_CSR_32));
  check(nvgraphAllocateVertexData(handle, graph, vertex_numsets, vertex_dimT));
  for (int i = 0; i < vertex_numsets; ++i) {
     check(nvgraphSetVertexData(handle, graph, vertex_dim[i], i));
  }
  check(nvgraphAllocateEdgeData(handle, graph, edge_numsets, &edge_dimT));
  check(nvgraphSetEdgeData(handle, graph, (void*)(&g.weights[0]), 0));

  // Solve

  check(nvgraphSrSpmv(handle, graph, 0, alpha_p, 0, beta_p, 1, NVGRAPH_MIN_PLUS_SR));
  hipDeviceSynchronize();

  stopwatch.start();
  check(nvgraphSrSpmv(handle, graph, 0, alpha_p, 0, beta_p, 1, NVGRAPH_MIN_PLUS_SR));
  hipDeviceSynchronize();
  stopwatch.stop();

  std::cout << "SrSpmv time: " << stopwatch.elapsed() << std::endl;
 
  //Get result
  check(nvgraphGetVertexData(handle, graph, (void*)(&y_h[0]), 1));

  //Clean 
  check(nvgraphDestroyGraphDescr(handle, graph));
  check(nvgraphDestroy(handle));
  free(vertex_dim); 
  free(vertex_dimT); 
  free(CSR_input);

}

void nvgraph_sssp_test() {

  size_t vertex_numsets = 1;
  size_t edge_numsets = 1;

  // nvgraph variables
  nvgraphHandle_t handle;
  nvgraphGraphDescr_t graph;
  nvgraphCSCTopology32I_t CSC_input;
  hipDataType edge_dimT = HIP_R_32F;

  std::vector < float > sssp_h(g.num_vertices);

  // Init host data
  CSC_input = (nvgraphCSCTopology32I_t) malloc(sizeof(struct nvgraphCSCTopology32I_st));

  void * vertex_dim[] = {(void*)(&sssp_h[0])}; 
  hipDataType vertex_dimT[] = {HIP_R_32F};

  check(nvgraphCreate(&handle));
  check(nvgraphCreateGraphDescr (handle, &graph));
  CSC_input->nvertices = g.num_vertices; 
  CSC_input->nedges = g.num_edges;
  CSC_input->source_indices = &g.destinations[0];
  CSC_input->destination_offsets = &g.offsets[0];

  // Set graph connectivity and properties (tranfers)
  check(nvgraphSetGraphStructure(handle, graph, (void*)CSC_input, NVGRAPH_CSC_32));
  check(nvgraphAllocateVertexData(handle, graph, vertex_numsets, vertex_dimT));
  check(nvgraphAllocateEdgeData  (handle, graph, edge_numsets, &edge_dimT));
  check(nvgraphSetEdgeData(handle, graph, (void*)(&g.weights[0]), 0));

  // Solve
  int source_vert = 0;
  check(nvgraphSssp(handle, graph, 0,  &source_vert, 0));
  hipDeviceSynchronize();

  stopwatch.start();
  check(nvgraphSssp(handle, graph, 0,  &source_vert, 0));
  hipDeviceSynchronize();
  stopwatch.stop();

  std::cout << "sssp time: " << stopwatch.elapsed() << std::endl;

  // Get and print result
  check(nvgraphGetVertexData(handle, graph, (void*)(&sssp_h[0]), 0));

  //Clean
  free(CSC_input);
  check(nvgraphDestroyGraphDescr(handle, graph));
  check(nvgraphDestroy(handle));

}

int main() {

  auto edges = read_binary< Graph::edge >(filename);

  g = Graph(edges);

  nvgraph_spmv_test();
  nvgraph_sssp_test();

}
