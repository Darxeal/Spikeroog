#include "hip/hip_runtime.h"
#include "misc/io.h"
#include "misc/timer.h"
#include "misc/graph.h"

#include "linear_algebra/math.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

std::string path("C:\\Users\\sam\\Desktop\\LUT\\");
std::string filename("C:\\Users\\sam\\Desktop\\LUT\\tuples.bin");

int main() {

  timer stopwatch;

  auto edges = read_binary< Graph::edge >(filename);
  auto points = read_binary< vec3 >(path + std::string("navigation_nodes.bin"));

  Graph g(edges);

  std::cout << std::endl;
  std::cout << g.num_vertices << " " << g.num_edges << std::endl;

  auto parents_dijk = g.dijkstra_sssp(72);
  auto parents_bf = g.bellman_ford_sssp(72);

  int errors = 0;
  for (int i = 0; i < g.num_vertices; i++) {
    if (parents_dijk[i] != parents_bf[i]) {
      errors++;
    }
  }

  std::cout << "found " << errors << " mismatches between dijkstra_sssp() and bellman_ford_sssp()" << std::endl;

  write_binary(parents_dijk, "C:\\Users\\sam\\Desktop\\LUT\\parents_dijk.bin");
  write_binary(parents_bf, "C:\\Users\\sam\\Desktop\\LUT\\parents_bf.bin");

  std::vector< bool > mask(g.num_vertices, false);

  float r = 4000.0f;
  vec3 p{0.0f, 0.0f, 0.0f};

  stopwatch.start();
  int count = 0;
  int closest = -1;
  float min_distance = 100000.0f;
  for (int i = 0; i < points.size(); i++) {
    float distance = norm(p - points[i]);
    if (distance < r) {
      for (int j = 0; j < 16; j++) {
        count++;
        mask[i * 16 + j] = true;
      }
    }

    if (distance < min_distance) {
      closest = i;
      min_distance = distance;
    }
  }
  stopwatch.stop();

  std::cout << count << " " << closest << " " << stopwatch.elapsed() << std::endl;

  stopwatch.start();
  parents_bf = g.bellman_ford_sssp(closest * 16 + 6);
  stopwatch.stop();
  std::cout << "bellman_ford_sssp() time: " << stopwatch.elapsed() << std::endl;

  return 0;

}

